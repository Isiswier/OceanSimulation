#include "fft.h"

/**************************************************
 * Assert: sizeof(complex) == sizeof(hipfftComplex)
 **************************************************
 */


void FFT::gpufft2(complex* in)
{
	// Copy host memory to device
	hipMemcpy(deviceData, in, memSize, hipMemcpyHostToDevice);


	// Transform 
	checkCudaErrors(hipfftExecC2C(plan, (hipfftComplex *)deviceData, (hipfftComplex *)deviceData, HIPFFT_FORWARD));


	// Check if kernel execution generated and error
	//getLastCudaError("Kernel execution failed");


	// Copy device memory to host
	checkCudaErrors(hipMemcpy(in, deviceData, memSize, hipMemcpyDeviceToHost));

}